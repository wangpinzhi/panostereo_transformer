#include "hip/hip_runtime.h"
/*!
 ******************* BEGIN Caffe Copyright Notice and Disclaimer ****************
 *
 * COPYRIGHT
 *
 * All contributions by the University of California:
 * Copyright (c) 2014-2017 The Regents of the University of California (Regents)
 * All rights reserved.
 *
 * All other contributions:
 * Copyright (c) 2014-2017, the respective contributors
 * All rights reserved.
 *
 * Caffe uses a shared copyright model: each contributor holds copyright over
 * their contributions to Caffe. The project versioning records all such
 * contribution and copyright details. If a contributor wants to further mark
 * their specific copyright on a particular contribution, they should indicate
 * their copyright solely in the commit message of the change when it is
 * committed.
 *
 * LICENSE
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
 * ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * CONTRIBUTION AGREEMENT
 *
 * By contributing to the BVLC/caffe repository through pull-request, comment,
 * or otherwise, the contributor releases their content to the
 * license and copyright terms herein.
 *
 ***************** END Caffe Copyright Notice and Disclaimer ********************
 *
 * Copyright (c) 2018 Microsoft
 * Licensed under The MIT License [see LICENSE for details]
 * \file modulated_deformable_im2col.cuh
 * \brief Function definitions of converting an image to
 * column matrix based on kernel, padding, dilation, and offset.
 * These functions are mainly used in deformable convolution operators.
 * \ref: https://arxiv.org/abs/1703.06211
 * \author Yuwen Xiong, Haozhi Qi, Jifeng Dai, Xizhou Zhu, Han Hu, Dazhi Cheng
 */

// modified from https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/blob/mmdetection/mmdet/ops/dcn/src/deform_conv_cuda_kernel.cu

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCAtomics.cuh>
#include <float.h>
#include <math.h>
#include <stdio.h>

using namespace at;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
const int kMaxGridNum = 65535;

inline int GET_BLOCKS(const int N) {
  return std::min(kMaxGridNum, (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS);
}

template <typename scalar_t>
__device__ scalar_t im2col_bilinear_sampling(const scalar_t * bottom_data, const int data_width,
                                             const int height, const int width, scalar_t h, scalar_t w) {

  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  scalar_t lh = h - h_low;
  scalar_t lw = w - w_low;
  scalar_t hh = 1 - lh, hw = 1 - lw;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
    v1 = bottom_data[h_low * data_width + w_low];
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = bottom_data[h_low * data_width + w_high];
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = bottom_data[h_high * data_width + w_low];
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = bottom_data[h_high * data_width + w_high];

  scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}
template <typename scalar_t>
__device__ scalar_t im2col_nearest_sampling(const scalar_t * bottom_data, const int data_width,
                                            const int height, const int width, scalar_t h, scalar_t w) {

  int hi = round(h);
  int wi = round(w);
  if (hi < 0) hi = 0;
  if (hi >= height) hi = height - 1;
  if (wi < 0) wi = 0;
  if (wi >= width) wi = width - 1;
  scalar_t val = bottom_data[hi * data_width + wi];
  return val;
}

template <typename scalar_t>
__device__ scalar_t get_gradient_weight(scalar_t argmax_h, scalar_t argmax_w,
                                        const int h, const int w, const int height, const int width) {

  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 || argmax_w >= width) {
    //empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
    weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
    weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
    weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
    weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

template <typename scalar_t>
__device__ scalar_t get_coordinate_weight(scalar_t argmax_h, scalar_t argmax_w,
                                          const int height, const int width, const scalar_t * im_data,
                                          const int data_width, const int bp_dir) {

  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 || argmax_w >= width) {
    //empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;

  if (bp_dir == 0) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_w_low + 1 - argmax_w) * im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += -1 * (argmax_w - argmax_w_low) * im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += (argmax_w_low + 1 - argmax_w) * im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_w - argmax_w_low) * im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_h_low + 1 - argmax_h) * im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += (argmax_h_low + 1 - argmax_h) * im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += -1 * (argmax_h - argmax_h_low) * im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_h - argmax_h_low) * im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

//data_position: B/col_step x col_step x (2 x Kh x Kw) x H x W
template <typename scalar_t>
__global__ void sphere_im2col_gpu_kernel(const int n,
                                         const scalar_t * data_im, const scalar_t * data_position,
                                         const int height, const int width, const int kernel_h, const int kernel_w,
                                         const int pad_h, const int pad_w,
                                         const int stride_h, const int stride_w,
                                         const int dilation_h, const int dilation_w,
                                         const int channel_per_deformable_group,
                                         const int batch_size, const int num_channels,
                                         const int height_col, const int width_col,
                                         scalar_t * data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    // index index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index - unused in sphere
    //const int deformable_group_index = c_im / channel_per_deformable_group;

    // const int h_in = h_col * stride_h - pad_h;
    // const int w_in = w_col * stride_w - pad_w; //
    // const int h_in_mid = h_in + pad_h;
    // const int w_in_mid = w_in + pad_w;
    const int h_in_mid = h_col * stride_h;
    const int w_in_mid = w_col * stride_w;

    scalar_t * data_col_ptr = data_col + ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    const scalar_t * data_im_ptr = data_im + ((b_col * num_channels + c_im) * height + 0) * width + 0;
    const scalar_t * data_position_ptr = data_position;
    //const scalar_t * data_im_ptr = data_im + (b_col * num_channels + c_im) * height * width;
    //const scalar_t * data_position_ptr = data_position + b_col * 2 * kernel_h * kernel_w * height_col * width_col;

    //const scalar_t * data_mask_ptr = data_mask + b_col * kernel_h * kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        // const int data_position_h_ptr = ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        // const int data_position_w_ptr = ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col + w_col;
        const int data_position_h_ptr = ((2 * (i * kernel_w + j)) * height + h_in_mid) * width + w_in_mid;
        const int data_position_w_ptr = ((2 * (i * kernel_w + j) + 1) * height + h_in_mid) * width + w_in_mid;
        //const int data_mask_hw_ptr = ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
        // const scalar_t position_h = data_position_ptr[data_position_h_ptr];
        // const scalar_t position_w = data_position_ptr[data_position_w_ptr];
        //const scalar_t mask = data_mask_ptr[data_mask_hw_ptr];
        scalar_t val = static_cast<scalar_t>(0);
        const scalar_t h_im = data_position_ptr[data_position_h_ptr];
        const scalar_t w_im = data_position_ptr[data_position_w_ptr];
        //if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) {
        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width) {
          //const float map_h = i * dilation_h + offset_h;
          //const float map_w = j * dilation_w + offset_w;
          //const int cur_height = height - h_in;
          //const int cur_width = width - w_in;
          //val = dmcn_im2col_bilinear(data_im_ptr, width, cur_height, cur_width, map_h, map_w);
          val = im2col_bilinear_sampling(data_im_ptr, width, height, width, h_im, w_im);
        }
        //if (h_col == 5 && w_col == 3) printf("im2col gpu kernel info: \nd_in: %d,%d; d_out: %d,%d; kernel (%d,%d) at (%f,%f),val=%f\n", h_in_mid, w_in_mid, h_col, w_col, i, j, h_im, w_im, val);
        // *data_col_ptr = val * mask;
        *data_col_ptr = val;
        data_col_ptr += batch_size * height_col * width_col;
        //data_col_ptr += height_col * width_col;
      }
    }
  }
}
void sphere_im2col_cuda(
  const at::Tensor data_im, const at::Tensor data_position,
  const int batch_size, const int channels, const int height_im, const int width_im,
  const int height_col, const int width_col, const int kernel_h, const int kenerl_w,
  const int pad_h, const int pad_w, const int stride_h, const int stride_w,
  const int dilation_h, const int dilation_w, at::Tensor data_col) {
  // num_axes should be smaller than block size
  const int channel_per_deformable_group = channels;
  const int num_kernels = channels * batch_size * height_col * width_col;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    data_im.scalar_type(), "sphere_im2col_gpu", ([&] {
      const scalar_t * data_im_ = data_im.data<scalar_t>();
      const scalar_t * data_position_ = data_position.data<scalar_t>();
      // const scalar_t * data_mask_ = data_mask.data<scalar_t>();
      scalar_t * data_col_ = data_col.data<scalar_t>();

      sphere_im2col_gpu_kernel<<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels, data_im_, data_position_, height_im, width_im, kernel_h, kenerl_w,
        pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w, channel_per_deformable_group,
        batch_size, channels, height_col, width_col, data_col_);
    }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in sphere_im2col_cuda: %s\n", hipGetErrorString(err));
  }
}

// col2im
template <typename scalar_t>
__global__ void sphere_col2im_gpu_kernel(const int n,
                                         const scalar_t * data_col, const scalar_t * data_position,
                                         const int channels, const int height, const int width,
                                         const int kernel_h, const int kernel_w,
                                         const int pad_h, const int pad_w,
                                         const int stride_h, const int stride_w,
                                         const int dilation_h, const int dilation_w,
                                         const int channel_per_deformable_group,
                                         const int batch_size,
                                         const int height_col, const int width_col,
                                         scalar_t * grad_im) {
  CUDA_KERNEL_LOOP(index, n) {
    const int j = (index / width_col / height_col / batch_size) % kernel_w;
    const int i = (index / width_col / height_col / batch_size / kernel_w) % kernel_h;
    const int c = index / width_col / height_col / batch_size / kernel_w / kernel_h;
    // compute the start and end of the output
    //unused in sphere
    //const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = index % width_col;
    int h_out = (index / width_col) % height_col;
    int b = (index / width_col / height_col) % batch_size;
    // int w_in = w_out * stride_w - pad_w;
    // int h_in = h_out * stride_h - pad_h;
    // const int h_in_mid = h_in + pad_h;
    // const int w_in_mid = w_in + pad_w;
    const int h_in_mid = h_out * stride_h;
    const int w_in_mid = w_out * stride_w;

    // const scalar_t * data_position_ptr = data_position + b * 2 * kernel_h * kernel_w * height_col * width_col;
    const scalar_t * data_position_ptr = data_position;
    //const scalar_t * data_mask_ptr = data_mask + b * kernel_h * kernel_w * height_col * width_col;
    // const int data_position_h_ptr = ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    // const int data_position_w_ptr = ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const int data_position_h_ptr = ((2 * (i * kernel_w + j)) * height + h_in_mid) * width + w_in_mid;
    const int data_position_w_ptr = ((2 * (i * kernel_w + j) + 1) * height + h_in_mid) * width + w_in_mid;
    //const int data_mask_hw_ptr = ((i * kernel_w + j) * height_col + h_out) * width_col + w_out;
    // const scalar_t position_h = data_position_ptr[data_position_h_ptr];
    // const scalar_t position_w = data_position_ptr[data_position_w_ptr];
    //const scalar_t mask = data_mask_ptr[data_mask_hw_ptr];
    const scalar_t cur_inv_h_data = data_position_ptr[data_position_h_ptr];
    const scalar_t cur_inv_w_data = data_position_ptr[data_position_w_ptr];
    const scalar_t cur_top_grad = data_col[index];
    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;
    //if (h_out == 5 && w_out == 3) printf("col2im gpu kernel info: \nd_in: %d,%d; d_out: %d,%d; at (%f,%f); int (%d,%d)\n", h_in_mid, w_in_mid, h_out, w_out, cur_inv_h_data, cur_inv_w_data, cur_h, cur_w);
    // int count = 0;
    for (int dy = 0; dy <= 1; dy++) {
      for (int dx = 0; dx <= 1; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height &&
            cur_w + dx >= 0 && cur_w + dx < width &&
            abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos = ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
          scalar_t weight = get_gradient_weight(cur_inv_h_data, cur_inv_w_data, cur_h + dy, cur_w + dx, height, width);
          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
          //if (h_out == 31 && w_out == 15) ++count;
        }
      }
    }
    //if (h_out == 31 && w_out == 15) printf("count: %d\n", count);
  }
}
void sphere_col2im_cuda(
  const at::Tensor data_col, const at::Tensor data_position,
  const int batch_size, const int channels, const int height_im, const int width_im,
  const int height_col, const int width_col, const int kernel_h, const int kernel_w,
  const int pad_h, const int pad_w, const int stride_h, const int stride_w,
  const int dilation_h, const int dilation_w, at::Tensor grad_im) {

  const int channel_per_deformable_group = channels;
  const int num_kernels = channels * kernel_h * kernel_w * batch_size * height_col * width_col;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    data_col.scalar_type(), "sphere_col2im_gpu", ([&] {
      const scalar_t * data_col_ = data_col.data<scalar_t>();
      const scalar_t * data_position_ = data_position.data<scalar_t>();
      //const scalar_t * data_mask_ = data_mask.data<scalar_t>();
      scalar_t * grad_im_ = grad_im.data<scalar_t>();

      sphere_col2im_gpu_kernel<<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
        num_kernels, data_col_, data_position_, channels, height_im, width_im,
        kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w,
        dilation_h, dilation_w, channel_per_deformable_group,
        batch_size, height_col, width_col, grad_im_);
    }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in sphere_col2im_cuda: %s\n", hipGetErrorString(err));
  }
}
